#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

static int N = 1000000;

#define CHECK(a) {err = hipGetLastError();\
if( err != hipSuccess ) \
printf(a);}

__global__ void SomeKernel(float *a, float *b, float *c, int nN)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < nN)
	{
		if (threadIdx.x%2 == 0) {
			c[id] = sin(a[id]) + sin(b[id]);
		}
		else {
			c[id] = sin(a[id]) - sin(b[id]);
		}
		
	}
}

void HostSum(float *a, float *b, float *c, int nN) {
	for (int i = 0; i < N; i++) {
		c[i] = sin(a[i]) + sin(b[i]);
	}
}

int main()
{
	hipError_t err;
	clock_t start, stop;
	float *h_a, *h_b, *h_res;
	float *d_a, *d_b, *d_res;

	//�������
	hipEvent_t event_start, event_stop, all_start, all_stop;
	float kernelTime = 0.0, allTime = 0.0;
	//������ �������
	hipEventCreate(&event_start);
	hipEventCreate(&event_stop);
	hipEventCreate(&all_start);
	hipEventCreate(&all_stop);



	// ��������� ����������� ������ (��� CPU)
	hipHostMalloc((void**)&h_a, N * sizeof(N));
	hipHostMalloc((void**)&h_b, N * sizeof(N));
	hipHostMalloc((void**)&h_res, N * sizeof(N));

	// ������������� �������� ������
	for (int i = 0; i < N; ++i)
	{
		h_a[i] = i / (i + 1.0);
		h_b[i] = i / (i + 3.0);
		h_res[i] = 0;
	}
	start = clock();
	HostSum(h_a, h_b, h_res, N);
	stop = clock();
	printf("Time Host sum = %d\n", (stop - start));

	
	// ��������� ������ GPU
	hipMalloc((void**)&d_a, N * sizeof(float));
	CHECK("cuda malloc 1\n");
	hipMalloc((void**)&d_b, N * sizeof(float));
	CHECK("cuda malloc 2\n");
	hipMalloc((void**)&d_res, N * sizeof(float));
	CHECK("cuda malloc 3\n");
	
	hipEventRecord(all_start, 0);
	hipMemcpyAsync(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice, 0);
	CHECK("cuda memcpy HostToDevice 1\n");
	hipMemcpyAsync(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice, 0);
	CHECK("cuda memcpy HostToDevice 2\n");

	int threads = 32;
	int blocks = N / 32;


	
	SomeKernel << <blocks, threads >> > (d_a, d_b, d_res, N);
	CHECK("kernel \n");
	
	hipMemcpyAsync(h_res, d_res, N * sizeof(float), hipMemcpyDeviceToHost, 0);
	CHECK("cuda memcpy DeviceToHost \n");


	hipEventRecord(all_stop, 0);
	hipEventSynchronize(all_stop);
	hipEventElapsedTime(&allTime, all_start, all_stop);
	printf("All time Device sum = %f\n", allTime);


	hipEventDestroy(all_start);
	hipEventDestroy(all_stop);
	hipEventDestroy(event_start);
	hipEventDestroy(event_stop);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_res);
	return 0;
}