#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

static int N = 1000000;

#define CHECK(a) {err = hipGetLastError();\
if( err != hipSuccess ) \
printf(a);}

__global__ void SomeKernel(float *a, float *b, float *c, int lN)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < lN)
	{
		c[id] = a[id] + b[id];
	}
}

void HostSum(float *a, float *b, float *&c, int lN) {
	for (int i = 0; i < lN; i++) {
		c[i] = a[i] + b[i];
	}
}

int main()
{
	hipError_t err;
	clock_t start, stop;
	float *h_a, *h_b, *h_res;
	float *d_a, *d_b, *d_res;

	//�������
	hipEvent_t  all_start, all_stop;
	float  allTime = 0.0;
	//������ �������
	hipEventCreate(&all_start);
	hipEventCreate(&all_stop);

	hipStream_t t[2];


	// ��������� ����������� ������ (��� CPU)
	hipHostMalloc((void**)&h_a, N * sizeof(float));
	hipHostMalloc((void**)&h_b, N * sizeof(float));
	hipHostMalloc((void**)&h_res, N * sizeof(float));

	

	// ������������� �������� ������
	for (int i = 0; i < N; ++i)
	{
		
		h_a[i] = i / (i + 1.0);
		h_b[i] = i / (i + 3.0);
		h_res[i] = 0;
			
	}
	start = clock();
	HostSum(h_a, h_b, h_res, N);
	
	stop = clock();
	printf("Time Host sum = %d\n", (stop - start));

	// ��������� ������ GPU
	
	hipMalloc((void**)&d_a, N * sizeof(float));
	CHECK("cuda malloc 1\n");
	hipMalloc((void**)&d_b, N * sizeof(float));
	CHECK("cuda malloc 2\n");
	hipMalloc((void**)&d_res, N * sizeof(float));
	CHECK("cuda malloc 3\n");
	
	for (int i = 0; i < 2; i++) {
		hipStreamCreate(&t[i]);
	}

	hipEventRecord(all_start, 0);
	for (int i = 0; i < 2; i++) {
		hipMemcpyAsync(d_a + i*(N/2), h_a + i*(N/2), (N/2) * sizeof(float), hipMemcpyHostToDevice, t[i]);
	}
	CHECK("cuda memcpy HostToDevice 2\n");

	int threads = 32;
	int blocks = (N/2) / 32;

	for (int i = 0; i < 2; i++) {
		SomeKernel << <blocks, threads, 0, t[i] >> > (d_a + i*(N/2), d_b+i*(N/2), d_res+i*(N/2), N/2);
	}
	CHECK("kernel \n");

	for (int i = 0; i < 2; i++) {
		hipMemcpyAsync(h_res + i*(N/2), d_res + i*(N/2), (N/2) * sizeof(float), hipMemcpyDeviceToHost, t[i]);
	}
	CHECK("cuda memcpy DeviceToHost \n");

	hipEventRecord(all_stop, 0);
	hipEventSynchronize(all_stop);
	hipEventElapsedTime(&allTime, all_start, all_stop);
	
	printf("All time Device sum = %f\n", allTime);


	hipEventDestroy(all_start);
	hipEventDestroy(all_stop);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_res);
	return 0;
}